#include "hip/hip_runtime.h"
﻿#include <GL/glew.h>
#include <GL/gl.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "SimulationSpace.h"
#include "Molecule.cuh"
#include "Atom.cuh"
#include "kernel.cuh"
#include "visualization.h"

// Define constants
#define MAX_MOLECULES 1000000
#define MAX_MOLECULE_TYPES 33
#define NUM_REACTION_TYPES 10 // Update this as you add more reaction types

// Constants for force calculations
#define COULOMB_CONSTANT 8.99e9f  // N*m^2/C^2
#define CUTOFF_DISTANCE 2.0f      // nm
#define CUTOFF_DISTANCE_SQ (CUTOFF_DISTANCE * CUTOFF_DISTANCE)
#define EPSILON_0 8.854187817e-12f // Vacuum permittivity
#define K_BOLTZMANN 1.380649e-23f  // Boltzmann constant
#define TEMPERATURE 310.15f        // Temperature in Kelvin (37°C)
#define SOLVENT_DIELECTRIC 78.5f   // Dielectric constant of water at 37°C

// Update these constants at the top of the file
#define MAX_THREADS_PER_BLOCK 1024
#define MAX_BLOCKS 65535

// Define window and isPaused
GLFWwindow* window;
bool isPaused = false;

// Function prototypes
__device__ float3 calculatePairwiseForce(const Atom& atom1, const Atom& atom2, float invDist, float distSq);
__global__ void calculateForces(Molecule* molecules, int num_molecules, float3* forces);
__global__ void applyForcesAndUpdatePositions(Molecule* molecules, float3* forces, int num_molecules, SimulationSpace space, float dt);
hipError_t runSimulation(SimulationSpace* space, Molecule* molecules, int num_ticks);
hipError_t runSimulationStep(SimulationSpace* space, Molecule* molecules);

// Add this function before the main() function

const char* getMoleculeTypeName(MoleculeType type) {
    switch(type) {
        case GLUCOSE: return "GLUCOSE";
        case ATP: return "ATP";
        case ADP: return "ADP";
        case GLUCOSE_6_PHOSPHATE: return "GLUCOSE_6_PHOSPHATE";
        case FRUCTOSE_6_PHOSPHATE: return "FRUCTOSE_6_PHOSPHATE";
        case FRUCTOSE_1_6_BISPHOSPHATE: return "FRUCTOSE_1_6_BISPHOSPHATE";
        case DIHYDROXYACETONE_PHOSPHATE: return "DIHYDROXYACETONE_PHOSPHATE";
        case GLYCERALDEHYDE_3_PHOSPHATE: return "GLYCERALDEHYDE_3_PHOSPHATE";
        case _1_3_BISPHOSPHOGLYCERATE: return "1_3_BISPHOSPHOGLYCERATE";
        case _3_PHOSPHOGLYCERATE: return "3_PHOSPHOGLYCERATE";
        case _2_PHOSPHOGLYCERATE: return "2_PHOSPHOGLYCERATE";
        case PHOSPHOENOLPYRUVATE: return "PHOSPHOENOLPYRUVATE";
        case PYRUVATE: return "PYRUVATE";
        case NAD_PLUS: return "NAD_PLUS";
        case NADH: return "NADH";
        case PROTON: return "PROTON";
        case INORGANIC_PHOSPHATE: return "INORGANIC_PHOSPHATE";
        case WATER: return "WATER";
        case HEXOKINASE: return "HEXOKINASE";
        case GLUCOSE_6_PHOSPHATE_ISOMERASE: return "GLUCOSE_6_PHOSPHATE_ISOMERASE";
        case PHOSPHOFRUCTOKINASE_1: return "PHOSPHOFRUCTOKINASE_1";
        case ALDOLASE: return "ALDOLASE";
        case TRIOSEPHOSPHATE_ISOMERASE: return "TRIOSEPHOSPHATE_ISOMERASE";
        case GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE: return "GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE";
        case PHOSPHOGLYCERATE_KINASE: return "PHOSPHOGLYCERATE_KINASE";
        case PHOSPHOGLYCERATE_MUTASE: return "PHOSPHOGLYCERATE_MUTASE";
        case ENOLASE: return "ENOLASE";
        case PYRUVATE_KINASE: return "PYRUVATE_KINASE";
        case AMP: return "AMP";
        case CITRATE: return "CITRATE";
        case FRUCTOSE_2_6_BISPHOSPHATE: return "FRUCTOSE_2_6_BISPHOSPHATE";
        default: return "UNKNOWN";
    }
}

// Host function to create a molecule
Molecule createMolecule(MoleculeType type) {
    switch (type) {
        // Substrates and products
        case GLUCOSE:
            return Molecule::createGlucose();
        case ATP:
            return Molecule::createATP();
        case ADP:
            return Molecule::createADP();
        case GLUCOSE_6_PHOSPHATE:
            return Molecule::createGlucose6Phosphate();
        case FRUCTOSE_6_PHOSPHATE:
            return Molecule::createFructose6Phosphate();
        case FRUCTOSE_1_6_BISPHOSPHATE:
            return Molecule::createFructose16Bisphosphate();
        case DIHYDROXYACETONE_PHOSPHATE:
            return Molecule::createDihydroxyacetonePhosphate();
        case GLYCERALDEHYDE_3_PHOSPHATE:
            return Molecule::createGlyceraldehyde3Phosphate();
        case _1_3_BISPHOSPHOGLYCERATE:
            return Molecule::create13Bisphosphoglycerate();
        case _3_PHOSPHOGLYCERATE:
            return Molecule::create3Phosphoglycerate();
        case _2_PHOSPHOGLYCERATE:
            return Molecule::create2Phosphoglycerate();
        case PHOSPHOENOLPYRUVATE:
            return Molecule::createPhosphoenolpyruvate();
        case PYRUVATE:
            return Molecule::createPyruvate();
        case NAD_PLUS:
            return Molecule::createNADPlus();
        case NADH:
            return Molecule::createNADH();
        case PROTON:
            return Molecule::createProton();
        case INORGANIC_PHOSPHATE:
            return Molecule::createInorganicPhosphate();
        case WATER:
            return Molecule::createWater();

        // Enzymes
        case HEXOKINASE:
            return Molecule::createHexokinase();
        case GLUCOSE_6_PHOSPHATE_ISOMERASE:
            return Molecule::createGlucose6PhosphateIsomerase();
        case PHOSPHOFRUCTOKINASE_1:
            return Molecule::createPhosphofructokinase1();
        case ALDOLASE:
            return Molecule::createAldolase();
        case TRIOSEPHOSPHATE_ISOMERASE:
            return Molecule::createTriosephosphateIsomerase();
        case GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE:
            return Molecule::createGlyceraldehyde3PhosphateDehydrogenase();
        case PHOSPHOGLYCERATE_KINASE:
            return Molecule::createPhosphoglycerateKinase();
        case PHOSPHOGLYCERATE_MUTASE:
            return Molecule::createPhosphoglycerateMutase();
        case ENOLASE:
            return Molecule::createEnolase();
        case PYRUVATE_KINASE:
            return Molecule::createPyruvateKinase();

        // Regulatory molecules
        case AMP:
            return Molecule::createAMP();
        case CITRATE:
            return Molecule::createCitrate();
        case FRUCTOSE_2_6_BISPHOSPHATE:
            return Molecule::createFructose26Bisphosphate();

        default:
            fprintf(stderr, "Unknown molecule type\n");
            return Molecule::createWater();  // Default case, could also throw an exception
    }
}

// Host function to process creation and deletion flags
void processCreationDeletionFlags(Molecule* molecules, int* num_molecules, int max_molecules, 
                                  const MoleculeCreationInfo* creationBuffer, int numCreations,
                                  const int* deletionBuffer, int numDeletions) {
    // Process deletions
    for (int i = 0; i < numDeletions; i++) {
        int indexToDelete = deletionBuffer[i];
        if (indexToDelete < *num_molecules - 1) {
            molecules[indexToDelete] = molecules[*num_molecules - 1];
        }
        (*num_molecules)--;
    }

    // Process creations
    for (int i = 0; i < numCreations; i++) {
        if (*num_molecules < max_molecules) {
            Molecule newMolecule = createMolecule(creationBuffer[i].type);
            newMolecule.setPosition(creationBuffer[i].x, creationBuffer[i].y, creationBuffer[i].z);
            molecules[*num_molecules] = newMolecule;
            (*num_molecules)++;
        }
    }
}

// Modify the runSimulation function to use runSimulationStep
hipError_t runSimulation(SimulationSpace* space, Molecule* molecules, int num_ticks) {
    printf("Starting simulation with %d molecules for %d ticks\n", space->num_molecules, num_ticks);

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }
    printf("CUDA device set successfully\n");

    // Main simulation loop
    for (int tick = 0; tick < num_ticks; tick++) {
        printf("Starting tick %d\n", tick);
        
        cudaStatus = runSimulationStep(space, molecules);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Simulation step failed! Error: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        printf("Completed tick %d\n", tick);
    }

    printf("Simulation completed\n");

    return cudaStatus;
}

// Add the new runSimulationStep function
hipError_t runSimulationStep(SimulationSpace* space, Molecule* molecules) {
    static Molecule* dev_molecules = nullptr;
    static float3* dev_forces = nullptr;
    static hiprandState* dev_states = nullptr;
    static int* dev_reactionCounts = nullptr;
    static int* dev_num_molecules = nullptr;
    static MoleculeCreationInfo* dev_creationBuffer = nullptr;
    static int* dev_numCreations = nullptr;
    static int* dev_deletionBuffer = nullptr;
    static int* dev_numDeletions = nullptr;

    hipError_t cudaStatus;
    int threadsPerBlock = 256;
    int blocksPerGrid = min((space->num_molecules + threadsPerBlock - 1) / threadsPerBlock, MAX_BLOCKS);

    // Initialize device memory if it's the first run
    if (dev_molecules == nullptr) {
        // Allocate GPU buffers and copy data
        cudaStatus = hipMalloc((void**)&dev_molecules, MAX_MOLECULES * sizeof(Molecule));
        cudaStatus = hipMalloc((void**)&dev_forces, MAX_MOLECULES * sizeof(float3));
        cudaStatus = hipMalloc((void**)&dev_states, MAX_MOLECULES * sizeof(hiprandState));
        cudaStatus = hipMalloc(&dev_reactionCounts, NUM_REACTION_TYPES * sizeof(int));
        cudaStatus = hipMalloc((void**)&dev_num_molecules, sizeof(int));
        cudaStatus = hipMalloc((void**)&dev_creationBuffer, MAX_MOLECULES * sizeof(MoleculeCreationInfo));
        cudaStatus = hipMalloc((void**)&dev_numCreations, sizeof(int));
        cudaStatus = hipMalloc((void**)&dev_deletionBuffer, MAX_MOLECULES * sizeof(int));
        cudaStatus = hipMalloc((void**)&dev_numDeletions, sizeof(int));

        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!\n");
            return cudaStatus;
        }

        // Initialize hiprandState
        initCurand<<<blocksPerGrid, threadsPerBlock>>>(time(NULL), dev_states, space->num_molecules);
    }

    // Copy current state to device
    cudaStatus = hipMemcpy(dev_molecules, molecules, space->num_molecules * sizeof(Molecule), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_num_molecules, &space->num_molecules, sizeof(int), hipMemcpyHostToDevice);
    hipMemset(dev_reactionCounts, 0, NUM_REACTION_TYPES * sizeof(int));
    hipMemset(dev_numCreations, 0, sizeof(int));
    hipMemset(dev_numDeletions, 0, sizeof(int));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    // Run kernels
    calculateForces<<<blocksPerGrid, threadsPerBlock>>>(dev_molecules, space->num_molecules, dev_forces);
    applyForcesAndUpdatePositions<<<blocksPerGrid, threadsPerBlock>>>(dev_molecules, dev_forces, space->num_molecules, *space, 0.01f);
    handleInteractions<<<blocksPerGrid, threadsPerBlock>>>(dev_molecules, dev_num_molecules, MAX_MOLECULES, dev_states, dev_reactionCounts, dev_creationBuffer, dev_numCreations, dev_deletionBuffer, dev_numDeletions);

    // Process creation and deletion flags
    int h_numCreations, h_numDeletions;
    hipMemcpy(&h_numCreations, dev_numCreations, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_numDeletions, dev_numDeletions, sizeof(int), hipMemcpyDeviceToHost);

    std::vector<MoleculeCreationInfo> h_creationBuffer(h_numCreations);
    std::vector<int> h_deletionBuffer(h_numDeletions);
    hipMemcpy(h_creationBuffer.data(), dev_creationBuffer, h_numCreations * sizeof(MoleculeCreationInfo), hipMemcpyDeviceToHost);
    hipMemcpy(h_deletionBuffer.data(), dev_deletionBuffer, h_numDeletions * sizeof(int), hipMemcpyDeviceToHost);

    processCreationDeletionFlags(molecules, &space->num_molecules, MAX_MOLECULES, h_creationBuffer.data(), h_numCreations, h_deletionBuffer.data(), h_numDeletions);

    // Copy results back to device
    cudaStatus = hipMemcpy(dev_molecules, molecules, space->num_molecules * sizeof(Molecule), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_num_molecules, &space->num_molecules, sizeof(int), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    // Check for errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernels!\n", cudaStatus);
        return cudaStatus;
    }

    // Copy results back to host
    cudaStatus = hipMemcpy(molecules, dev_molecules, space->num_molecules * sizeof(Molecule), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    return cudaStatus;
}

// Main function
int main() {
    // Get and print CUDA device properties
    hipDeviceProp_t deviceProp;
    hipError_t cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed! Error: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    printf("CUDA Device Properties:\n");
    printf("  Device name: %s\n", deviceProp.name);
    printf("  Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("  Total global memory: %zu bytes\n", deviceProp.totalGlobalMem);
    printf("  Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    printf("  Max threads dim: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("  Max grid size: (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    printf("  Warp size: %d\n", deviceProp.warpSize);
    printf("  Memory clock rate: %d kHz\n", deviceProp.memoryClockRate);
    printf("  Memory bus width: %d bits\n", deviceProp.memoryBusWidth);
    printf("\n");

    SimulationSpace space;
    Molecule* molecules = nullptr;
    
    // Read input from file
    FILE* input_file = fopen("input.txt", "r");
    if (input_file == NULL) {
        fprintf(stderr, "Failed to open input.txt\n");
        return 1;
    }
    
    char line[256];
    
    // Read simulation space dimensions
    while (fgets(line, sizeof(line), input_file)) {
        if (line[0] != '#') {
            if (sscanf(line, "%d %d %d", &space.width, &space.height, &space.depth) != 3) {
                fprintf(stderr, "Failed to read simulation space dimensions\n");
                fclose(input_file);
                return 1;
            }
            printf("Simulation space dimensions: %d x %d x %d\n", space.width, space.height, space.depth);
            break;
        }
    }
    
    // Read number of molecule types
    while (fgets(line, sizeof(line), input_file)) {
        if (line[0] != '#') {
            if (sscanf(line, "%d", &space.num_molecule_types) != 1) {
                fprintf(stderr, "Failed to read number of molecule types\n");
                fclose(input_file);
                return 1;
            }
            printf("Number of molecule types: %d\n", space.num_molecule_types);
            break;
        }
    }
    
    // Read molecule counts
    space.num_molecules = 0;
    int molecule_type = 0;
    while (fgets(line, sizeof(line), input_file) && molecule_type < space.num_molecule_types) {
        if (line[0] != '#') {
            char molecule_name[50];
            int count;
            if (sscanf(line, "%[^:]: %d", molecule_name, &count) == 2) {
                space.molecule_counts[molecule_type] = count;
                space.num_molecules += count;
                printf("Molecule type %d (%s): %d\n", molecule_type, molecule_name, count);
                molecule_type++;
            }
        }
    }
    
    fclose(input_file);
    
    printf("Total number of molecules: %d\n", space.num_molecules);
    
    // Before allocating memory for molecules
    if (space.num_molecules > MAX_MOLECULES) {
        fprintf(stderr, "Error: Number of molecules (%d) exceeds maximum allowed (%d)\n", space.num_molecules, MAX_MOLECULES);
        return 1;
    }

    // Allocate memory for molecules
    molecules = (Molecule*)malloc(MAX_MOLECULES * sizeof(Molecule));
    if (molecules == nullptr) {
        fprintf(stderr, "Failed to allocate memory for molecules\n");
        return 1;
    }

    printf("Molecules allocated successfully\n");
    
    // Initialize molecules
    int molecule_index = 0;
    int type_counts[MAX_MOLECULE_TYPES] = {0};  // Array to keep count of each molecule type

    for (int i = 0; i < space.num_molecule_types; i++) {
        MoleculeType currentType = static_cast<MoleculeType>(i);
        int count = space.molecule_counts[i];
        
        printf("Creating %d molecules of type %d (%s)\n", count, i, getMoleculeTypeName(currentType));

        for (int j = 0; j < count; j++) {
            if (molecule_index >= MAX_MOLECULES) {
                fprintf(stderr, "Error: Exceeded maximum number of molecules (%d)\n", MAX_MOLECULES);
                free(molecules);
                return 1;
            }

            molecules[molecule_index] = createMolecule(currentType);

            // Initialize position randomly
            float x = static_cast<float>(rand()) / RAND_MAX * space.width;
            float y = static_cast<float>(rand()) / RAND_MAX * space.height;
            float z = static_cast<float>(rand()) / RAND_MAX * space.depth;
            molecules[molecule_index].setPosition(x, y, z);

            type_counts[i]++;
            molecule_index++;
        }
        printf("Created molecule type %d (%s): %d\n", i, getMoleculeTypeName(currentType), type_counts[i]);
    }

    // Print summary of all molecule types created
    printf("\nSummary of molecules created:\n");
    for (int i = 0; i < space.num_molecule_types; i++) {
        if (type_counts[i] > 0) {
            printf("%s: %d\n", getMoleculeTypeName(static_cast<MoleculeType>(i)), type_counts[i]);
        }
    }

    // Check if we've initialized the correct number of molecules
    if (molecule_index != space.num_molecules) {
        fprintf(stderr, "Error: Initialized %d molecules, expected %d\n", molecule_index, space.num_molecules);
        free(molecules);
        return 1;
    }

    printf("\nTotal molecules initialized successfully: %d\n", molecule_index);

    // Initialize visualization
    initVisualization();

    // Main simulation loop
    while (!glfwWindowShouldClose(window)) {
        if (!isPaused) {
            // Run a single step of the simulation
            cudaStatus = runSimulationStep(&space, molecules);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "Simulation step failed! Error: %s\n", hipGetErrorString(cudaStatus));
                break;
            }
        }

        // Render the current state of the simulation
        renderSimulation(space, std::vector<Molecule>(molecules, molecules + space.num_molecules));
    }

    // Cleanup
    cleanupVisualization();
    free(molecules);

    return 0;
}
// CUDA kernels
__device__ float3 calculatePairwiseForce(const Atom& atom1, const Atom& atom2, float invDist, float distSq) {
    float3 force = make_float3(0.0f, 0.0f, 0.0f);

    // Lennard-Jones potential
    float sigma = 0.5f * (atom1.getVanDerWaalsRadius() + atom2.getVanDerWaalsRadius());
    float epsilon = sqrtf(atom1.getEpsilon() * atom2.getEpsilon()); // Combining rule
    float sigmaOverDist = sigma * invDist;
    float sigmaOverDist6 = sigmaOverDist * sigmaOverDist * sigmaOverDist;
    sigmaOverDist6 = sigmaOverDist6 * sigmaOverDist6;
    float forceMultiplierLJ = 24.0f * epsilon * invDist * sigmaOverDist6 * (1.0f - 2.0f * sigmaOverDist6);

    // Coulomb force with Generalized Born model
    float fGB = sqrtf(distSq + atom1.getBornRadius() * atom2.getBornRadius() * expf(-distSq / (4.0f * atom1.getBornRadius() * atom2.getBornRadius())));
    float dielectric = 1.0f / (1.0f / SOLVENT_DIELECTRIC - 1.0f / 1.0f) * expf(-distSq / (4.0f * atom1.getBornRadius() * atom2.getBornRadius()));
    float forceMultiplierCoulomb = COULOMB_CONSTANT * atom1.getCharge() * atom2.getCharge() * (1.0f / distSq - 1.0f / (fGB * fGB)) / dielectric;

    float totalForceMultiplier = forceMultiplierLJ + forceMultiplierCoulomb;

    force.x = (atom2.getX() - atom1.getX()) * totalForceMultiplier * invDist;
    force.y = (atom2.getY() - atom1.getY()) * totalForceMultiplier * invDist;
    force.z = (atom2.getZ() - atom1.getZ()) * totalForceMultiplier * invDist;

    return force;
}