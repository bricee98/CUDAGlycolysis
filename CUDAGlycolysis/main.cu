#include "hip/hip_runtime.h"
﻿#include <GL/glew.h>
#include <GL/gl.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "SimulationSpace.h"
#include "Molecule.cuh"
#include "kernel.cuh"
#include "visualization.h"
#include "Cell.cuh"
#include "SimulationData.h"
#include <chrono>
#include <numeric>
#include <fstream>
#include <algorithm>
#include <vector>
#include <cassert>
#include <conio.h>
#include <cmath>

std::ofstream logFile("performance_log.txt");

// Define constants
#define MAX_MOLECULE_TYPES 33

// Constants for force calculations
#define COULOMB_CONSTANT 8.99e9f  // N*m^2/C^2
#define CUTOFF_DISTANCE 2.0f      // nm
#define CUTOFF_DISTANCE_SQ (CUTOFF_DISTANCE * CUTOFF_DISTANCE)
#define EPSILON_0 8.854187817e-12f // Vacuum permittivity
#define K_BOLTZMANN 1.380649e-23f  // Boltzmann constant
#define TEMPERATURE 310.15f        // Temperature in Kelvin (37°C)
#define SOLVENT_DIELECTRIC 78.5f   // Dielectric constant of water at 37°C

#define MAX_MOLECULES 6000

int g_threadsToUse;
int g_blocksToUse;

int recommended_max_molecules;
int recommended_max_molecules_per_cell;

// Add these declarations
extern int h_GRID_SIZE_X;
extern int h_GRID_SIZE_Y;
extern int h_GRID_SIZE_Z;

// Define window and isPaused
GLFWwindow* window;
bool isPaused = false;

float total_simulated_time = 0.0f;

// Add this global variable at the top of main.cu
bool isRenderingPaused = false;

// Add this global variable
bool printTimings = true;

// Function prototypes
hipError_t runSimulation(SimulationSpace* space, Molecule* molecules, int num_ticks);
hipError_t runSimulationStep(SimulationSpace* space, Molecule* molecules);

// Add this function before the main() function

const char* getMoleculeTypeName(MoleculeType type) {
    switch(type) {
        case GLUCOSE: return "GLUCOSE";
        case ATP: return "ATP";
        case ADP: return "ADP";
        case GLUCOSE_6_PHOSPHATE: return "GLUCOSE_6_PHOSPHATE";
        case FRUCTOSE_6_PHOSPHATE: return "FRUCTOSE_6_PHOSPHATE";
        case FRUCTOSE_1_6_BISPHOSPHATE: return "FRUCTOSE_1_6_BISPHOSPHATE";
        case DIHYDROXYACETONE_PHOSPHATE: return "DIHYDROXYACETONE_PHOSPHATE";
        case GLYCERALDEHYDE_3_PHOSPHATE: return "GLYCERALDEHYDE_3_PHOSPHATE";
        case _1_3_BISPHOSPHOGLYCERATE: return "1_3_BISPHOSPHOGLYCERATE";
        case _3_PHOSPHOGLYCERATE: return "3_PHOSPHOGLYCERATE";
        case _2_PHOSPHOGLYCERATE: return "2_PHOSPHOGLYCERATE";
        case PHOSPHOENOLPYRUVATE: return "PHOSPHOENOLPYRUVATE";
        case PYRUVATE: return "PYRUVATE";
        case NAD_PLUS: return "NAD_PLUS";
        case NADH: return "NADH";
        case PROTON: return "PROTON";
        case INORGANIC_PHOSPHATE: return "INORGANIC_PHOSPHATE";
        case WATER: return "WATER";
        case HEXOKINASE: return "HEXOKINASE";
        case PHOSPHOGLUCOSE_ISOMERASE: return "PHOSPHOGLUCOSE_ISOMERASE";
        case PHOSPHOFRUCTOKINASE_1: return "PHOSPHOFRUCTOKINASE_1";
        case ALDOLASE: return "ALDOLASE";
        case TRIOSEPHOSPHATE_ISOMERASE: return "TRIOSEPHOSPHATE_ISOMERASE";
        case GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE: return "GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE";
        case PHOSPHOGLYCERATE_KINASE: return "PHOSPHOGLYCERATE_KINASE";
        case PHOSPHOGLYCERATE_MUTASE: return "PHOSPHOGLYCERATE_MUTASE";
        case ENOLASE: return "ENOLASE";
        case PYRUVATE_KINASE: return "PYRUVATE_KINASE";
        case AMP: return "AMP";
        case CITRATE: return "CITRATE";
        case FRUCTOSE_2_6_BISPHOSPHATE: return "FRUCTOSE_2_6_BISPHOSPHATE";
        case HEXOKINASE_GLUCOSE_COMPLEX: return "HEXOKINASE_GLUCOSE_COMPLEX";
        case HEXOKINASE_GLUCOSE_ATP_COMPLEX: return "HEXOKINASE_GLUCOSE_ATP_COMPLEX";
        case GLUCOSE_6_PHOSPHATE_ISOMERASE_COMPLEX: return "GLUCOSE_6_PHOSPHATE_ISOMERASE_COMPLEX";
        case FRUCTOSE_6_PHOSPHATE_ISOMERASE_COMPLEX: return "FRUCTOSE_6_PHOSPHATE_ISOMERASE_COMPLEX";
        case PHOSPHOFRUCTOKINASE_1_COMPLEX: return "PHOSPHOFRUCTOKINASE_1_COMPLEX";
        case PHOSPHOFRUCTOKINASE_1_ATP_COMPLEX: return "PHOSPHOFRUCTOKINASE_1_ATP_COMPLEX";
        case FRUCTOSE_1_6_BISPHOSPHATE_ALDOLASE_COMPLEX: return "FRUCTOSE_1_6_BISPHOSPHATE_ALDOLASE_COMPLEX";
        case GLYCERALDEHYDE_3_PHOSPHATE_ALDOLASE_COMPLEX: return "GLYCERALDEHYDE_3_PHOSPHATE_ALDOLASE_COMPLEX";
        case GLYCERALDEHYDE_3_PHOSPHATE_ALDOLASE_DHAP_COMPLEX: return "GLYCERALDEHYDE_3_PHOSPHATE_ALDOLASE_DHAP_COMPLEX";
        case DHAP_TRIOSEPHOSPHATE_ISOMERASE_COMPLEX: return "DHAP_TRIOSEPHOSPHATE_ISOMERASE_COMPLEX";
        case GLYCERALDEHYDE_3_PHOSPHATE_TRIOSEPHOSPHATE_ISOMERASE_COMPLEX: return "GLYCERALDEHYDE_3_PHOSPHATE_TRIOSEPHOSPHATE_ISOMERASE_COMPLEX";
        case GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE_COMPLEX: return "GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE_COMPLEX";
        case GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE_NAD_PLUS_COMPLEX: return "GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE_NAD_PLUS_COMPLEX";
        case GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE_NAD_PLUS_INORGANIC_PHOSPHATE_COMPLEX: return "GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE_NAD_PLUS_INORGANIC_PHOSPHATE_COMPLEX";
        case PHOSPHOGLYCERATE_KINASE_COMPLEX: return "PHOSPHOGLYCERATE_KINASE_COMPLEX";
        case PHOSPHOGLYCERATE_KINASE_ADP_COMPLEX: return "PHOSPHOGLYCERATE_KINASE_ADP_COMPLEX";
        case PHOSPHOGLYCERATE_MUTASE_COMPLEX: return "PHOSPHOGLYCERATE_MUTASE_COMPLEX";
        case ENOLASE_COMPLEX: return "ENOLASE_COMPLEX";
        case PYRUVATE_KINASE_COMPLEX: return "PYRUVATE_KINASE_COMPLEX";
        case PYRUVATE_KINASE_ADP_COMPLEX: return "PYRUVATE_KINASE_ADP_COMPLEX";
        case NONE: return "NONE";
        default: return "UNKNOWN";
    }
}

// Host function to create a molecule
Molecule createMolecule(MoleculeType type) {
    switch (type) {
        // Substrates and products
        case GLUCOSE: return Molecule::createGlucose();
        case ATP: return Molecule::createATP();
        case ADP: return Molecule::createADP();
        case GLUCOSE_6_PHOSPHATE: return Molecule::createGlucose6Phosphate();
        case FRUCTOSE_6_PHOSPHATE: return Molecule::createFructose6Phosphate();
        case FRUCTOSE_1_6_BISPHOSPHATE: return Molecule::createFructose16Bisphosphate();
        case DIHYDROXYACETONE_PHOSPHATE: return Molecule::createDihydroxyacetonePhosphate();
        case GLYCERALDEHYDE_3_PHOSPHATE: return Molecule::createGlyceraldehyde3Phosphate();
        case _1_3_BISPHOSPHOGLYCERATE: return Molecule::create13Bisphosphoglycerate();
        case _3_PHOSPHOGLYCERATE: return Molecule::create3Phosphoglycerate();
        case _2_PHOSPHOGLYCERATE: return Molecule::create2Phosphoglycerate();
        case PHOSPHOENOLPYRUVATE: return Molecule::createPhosphoenolpyruvate();
        case PYRUVATE: return Molecule::createPyruvate();
        case NAD_PLUS: return Molecule::createNADPlus();
        case NADH: return Molecule::createNADH();
        case PROTON: return Molecule::createProton();
        case INORGANIC_PHOSPHATE: return Molecule::createInorganicPhosphate();
        case WATER: return Molecule::createWater();
        case AMP: return Molecule::createAMP();
        case CITRATE: return Molecule::createCitrate();
        case FRUCTOSE_2_6_BISPHOSPHATE: return Molecule::createFructose26Bisphosphate();

        // Enzymes
        case HEXOKINASE: return Molecule::createHexokinase();
        case PHOSPHOGLUCOSE_ISOMERASE: return Molecule::createGlucose6PhosphateIsomerase();
        case PHOSPHOFRUCTOKINASE_1: return Molecule::createPhosphofructokinase1();
        case ALDOLASE: return Molecule::createAldolase();
        case TRIOSEPHOSPHATE_ISOMERASE: return Molecule::createTriosephosphateIsomerase();
        case GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE: return Molecule::createGlyceraldehyde3PhosphateDehydrogenase();
        case PHOSPHOGLYCERATE_KINASE: return Molecule::createPhosphoglycerateKinase();
        case PHOSPHOGLYCERATE_MUTASE: return Molecule::createPhosphoglycerateMutase();
        case ENOLASE: return Molecule::createEnolase();
        case PYRUVATE_KINASE: return Molecule::createPyruvateKinase();

        // Enzyme complexes
        case HEXOKINASE_GLUCOSE_COMPLEX: return Molecule::createHexokinaseGlucoseComplex();
        case HEXOKINASE_GLUCOSE_ATP_COMPLEX: return Molecule::createHexokinaseGlucoseATPComplex();
        case GLUCOSE_6_PHOSPHATE_ISOMERASE_COMPLEX: return Molecule::createGlucose6PhosphateIsomeraseComplex();
        case FRUCTOSE_6_PHOSPHATE_ISOMERASE_COMPLEX: return Molecule::createFructose6PhosphateIsomeraseComplex();
        case PHOSPHOFRUCTOKINASE_1_COMPLEX: return Molecule::createPhosphofructokinase1Complex();
        case PHOSPHOFRUCTOKINASE_1_ATP_COMPLEX: return Molecule::createPhosphofructokinase1ATPComplex();
        case FRUCTOSE_1_6_BISPHOSPHATE_ALDOLASE_COMPLEX: return Molecule::createFructose16BisphosphateAldolaseComplex();
        case GLYCERALDEHYDE_3_PHOSPHATE_ALDOLASE_COMPLEX: return Molecule::createGlyceraldehyde3PhosphateAldolaseComplex();
        case GLYCERALDEHYDE_3_PHOSPHATE_ALDOLASE_DHAP_COMPLEX: return Molecule::createGlyceraldehyde3PhosphateAldolaseDHAPComplex();
        case DHAP_TRIOSEPHOSPHATE_ISOMERASE_COMPLEX: return Molecule::createDHAPTriosephosphateIsomeraseComplex();
        case GLYCERALDEHYDE_3_PHOSPHATE_TRIOSEPHOSPHATE_ISOMERASE_COMPLEX: return Molecule::createGlyceraldehyde3PhosphateTriosephosphateIsomeraseComplex();
        case GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE_COMPLEX: return Molecule::createGlyceraldehyde3PhosphateDehydrogenaseComplex();
        case GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE_NAD_PLUS_COMPLEX: return Molecule::createGlyceraldehyde3PhosphateDehydrogenaseNADPlusComplex();
        case GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE_NAD_PLUS_INORGANIC_PHOSPHATE_COMPLEX: return Molecule::createGlyceraldehyde3PhosphateDehydrogenaseNADPlusInorganicPhosphateComplex();
        case PHOSPHOGLYCERATE_KINASE_COMPLEX: return Molecule::createPhosphoglycerateKinaseComplex();
        case PHOSPHOGLYCERATE_KINASE_ADP_COMPLEX: return Molecule::createPhosphoglycerateKinaseADPComplex();
        case PHOSPHOGLYCERATE_MUTASE_COMPLEX: return Molecule::createPhosphoglycerateMutaseComplex();
        case ENOLASE_COMPLEX: return Molecule::createEnolaseComplex();
        case PYRUVATE_KINASE_COMPLEX: return Molecule::createPyruvateKinaseComplex();
        case PYRUVATE_KINASE_ADP_COMPLEX: return Molecule::createPyruvateKinaseADPComplex();

        case NONE: return Molecule::createNone();

        default:
            fprintf(stderr, "Unknown molecule type: %d\n", type);
            return Molecule::createWater();  // Default case, could also throw an exception
    }
}

// Host function to process creation and deletion flags
void processCreationDeletionFlags(Molecule* molecules, int* num_molecules, int max_molecules,
                                  const MoleculeCreationInfo* creationBuffer, int numCreations,
                                  const int* deletionBuffer, int numDeletions) {
    // Process deletions
    if (deletionBuffer && numDeletions > 0) {
        std::vector<int> deletionIndices(deletionBuffer, deletionBuffer + numDeletions);
        std::sort(deletionIndices.begin(), deletionIndices.end(), std::greater<int>());
        for (int i = 0; i < numDeletions; i++) {
            int indexToDelete = deletionIndices[i];
            assert(indexToDelete >= 0 && indexToDelete < *num_molecules); // Ensure valid index
            if (indexToDelete != *num_molecules - 1) {
                molecules[indexToDelete] = molecules[*num_molecules - 1];
            }
            (*num_molecules)--;
            //printf("Successfully deleted molecule at index %d\n", indexToDelete);
        }
    }

    // Process creations
    if (creationBuffer && numCreations > 0) {
        for (int i = 0; i < numCreations; i++) {
            assert(*num_molecules < max_molecules); // Ensure we don't exceed max molecules
            Molecule newMolecule = createMolecule(creationBuffer[i].type);
            //printf("Created molecule of type %s\n", getMoleculeTypeName(newMolecule.type));
            newMolecule.centerOfMass = make_float3(creationBuffer[i].x, creationBuffer[i].y, creationBuffer[i].z);
            molecules[*num_molecules] = newMolecule;
            (*num_molecules)++;
            //printf("Successfully added molecule to molecules array\n");
        }
    }
}

// Modify the runSimulation function to use runSimulationStep
hipError_t runSimulation(SimulationSpace* space, Molecule* molecules, int num_ticks) {
    assert(space != nullptr); // Ensure space is not null
    assert(molecules != nullptr); // Ensure molecules array is not null
    assert(num_ticks > 0); // Ensure there is at least one tick

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }
    //printf("CUDA device set successfully\n");

    // Main simulation loop
    for (int tick = 0; tick < num_ticks; tick++) {
        //printf("Starting tick %d\n", tick);

        cudaStatus = runSimulationStep(space, molecules);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Simulation step failed! Error: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        //printf("Completed tick %d\n", tick);
    }

    //printf("Simulation completed\n");

    return cudaStatus;
}

hipError_t runSimulationStep(SimulationSpace* space, Molecule* molecules) {
    static Molecule* dev_molecules = nullptr;
    static hiprandState* dev_states = nullptr;
    static int* dev_num_molecules = nullptr;
    static MoleculeCreationInfo* dev_creationBuffer = nullptr;
    static int* dev_numCreations = nullptr;
    static int* dev_deletionBuffer = nullptr;
    static int* dev_numDeletions = nullptr;
    static Cell* dev_cells = nullptr;

    // Remove the extern variables and define grid sizes locally
    int gridSizeX = static_cast<int>(space->width / CELL_SIZE);
    int gridSizeY = static_cast<int>(space->height / CELL_SIZE);
    int gridSizeZ = static_cast<int>(space->depth / CELL_SIZE);

    Grid grid;
    grid.sizeX = gridSizeX;
    grid.sizeY = gridSizeY;
    grid.sizeZ = gridSizeZ;

    hipError_t cudaStatus;
    int threadsPerBlock = g_threadsToUse;
    int blocksPerGrid = g_blocksToUse;

    // Calculate total cells
    int totalCells = grid.sizeX * grid.sizeY * grid.sizeZ;

    dim3 gridAssign(blocksPerGrid, 1, 1);

    // CUDA event creation for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //printf("Starting simulation step\n");

    // CPU timing variables
    std::chrono::high_resolution_clock::time_point t1, t2;

    // Add these new timing variables
    float resetCellsTime = 0.0f;
    float initCurandTime = 0.0f;
    float resetCreationDeletionBuffersTime = 0.0f;

    // Timing: Memory allocation and initialization
    t1 = std::chrono::high_resolution_clock::now();

    if (dev_cells == nullptr) {
        cudaStatus = hipMalloc(&dev_cells, totalCells * sizeof(Cell));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed for dev_cells!\n");
            return cudaStatus;
        }
    }

    // Timing: Reset cells
    hipEventRecord(start);
    hipMemset(dev_cells, 0, totalCells * sizeof(Cell));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&resetCellsTime, start, stop);

    // Initialize device memory if it's the first run
    if (dev_molecules == nullptr) {

        printf("Max molecules is %d\n", MAX_MOLECULES);
        printf("About to allocate max_molecules * sizeof(Molecule) = %zu\n", MAX_MOLECULES * sizeof(Molecule));

        // Allocate GPU buffers and copy data
        cudaStatus = hipMalloc((void**)&dev_molecules, MAX_MOLECULES * sizeof(Molecule));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed for dev_molecules! Error: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }
        printf("Successfully allocated dev_molecules with size %zu\n", MAX_MOLECULES * sizeof(Molecule));
        cudaStatus = hipMalloc((void**)&dev_states, blocksPerGrid * threadsPerBlock * sizeof(hiprandState));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed for dev_states!\n"); return cudaStatus; }
        printf("Successfully allocated dev_states with size %zu\n", blocksPerGrid * threadsPerBlock * sizeof(hiprandState));
        cudaStatus = hipMalloc((void**)&dev_num_molecules, sizeof(int));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed for dev_num_molecules!\n"); return cudaStatus; }
        printf("Successfully allocated dev_num_molecules with size %zu\n", sizeof(int));
        cudaStatus = hipMalloc((void**)&dev_creationBuffer, MAX_MOLECULES * sizeof(MoleculeCreationInfo));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed for dev_creationBuffer!\n"); return cudaStatus; }
        printf("Successfully allocated dev_creationBuffer with size %zu\n", MAX_MOLECULES * sizeof(MoleculeCreationInfo));
        cudaStatus = hipMalloc((void**)&dev_numCreations, sizeof(int));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed for dev_numCreations!\n"); return cudaStatus; }
        printf("Successfully allocated dev_numCreations with size %zu\n", sizeof(int));
        cudaStatus = hipMalloc((void**)&dev_deletionBuffer, MAX_MOLECULES * sizeof(int));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed for dev_deletionBuffer!\n"); return cudaStatus; }
        printf("Successfully allocated dev_deletionBuffer with size %zu\n", MAX_MOLECULES * sizeof(int));
        cudaStatus = hipMalloc((void**)&dev_numDeletions, sizeof(int));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed for dev_numDeletions!\n"); return cudaStatus; }
        printf("Successfully allocated dev_numDeletions with size %zu\n", sizeof(int));

        // Timing: Initialize hiprandState
        hipEventRecord(start);
        initCurand<<<blocksPerGrid, threadsPerBlock>>>(time(NULL), dev_states, blocksPerGrid * threadsPerBlock);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&initCurandTime, start, stop);
    }

    t2 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> memoryAllocationTime = t2 - t1;
    //printf("Successfully allocated memory\n");

    // Timing: Memory copy to device
    hipEventRecord(start);
    cudaStatus = hipMemcpy(dev_molecules, molecules, MAX_MOLECULES * sizeof(Molecule), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for molecules to dev_molecules! Error: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }
    //printf("Successfully copied molecules to device\n");

    cudaStatus = hipMemcpy(dev_num_molecules, &space->num_molecules, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for num_molecules to dev_num_molecules!\n");
        return cudaStatus;
    }
    //printf("Successfully copied num_molecules to device\n");

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float memcpyToDeviceTime;
    hipEventElapsedTime(&memcpyToDeviceTime, start, stop);

    // Timing: Reset the creation and deletion buffers
    hipEventRecord(start);
    hipMemset(dev_numCreations, 0, sizeof(int));
    hipMemset(dev_numDeletions, 0, sizeof(int));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&resetCreationDeletionBuffersTime, start, stop);

    //printf("Successfully set numCreations and numDeletions to 0\n");

    // Timing: Assign molecules to cells kernel
    hipEventRecord(start);
    //dim3 gridAssign((space->num_molecules + threadsPerBlock - 1) / threadsPerBlock);
    assignMoleculesToCells<<<blocksPerGrid, threadsPerBlock>>>(dev_molecules, space->num_molecules, dev_cells, *space, grid);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float assignMoleculesTime;
    hipEventElapsedTime(&assignMoleculesTime, start, stop);
    //printf("Successfully assigned molecules to cells\n");

    // Timing: Apply forces and update positions kernel
    hipEventRecord(start);
    float dt = 1e-6f; // Timestep of 1 microsecond
    applyForcesAndUpdatePositions<<<blocksPerGrid, threadsPerBlock>>>(
        dev_molecules, space->num_molecules, *space, dt, dev_states);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float applyForcesTime;
    hipEventElapsedTime(&applyForcesTime, start, stop);
    //printf("Successfully applied forces and updated positions\n");
    // Timing: handle binding reactions kernel
    hipEventRecord(start);
    handleBindings<<<blocksPerGrid, threadsPerBlock>>>(dev_molecules, dev_num_molecules, MAX_MOLECULES, dev_states,
                                                      dev_creationBuffer, dev_numCreations,
                                                      dev_deletionBuffer, dev_numDeletions,
                                                      dev_cells, grid);

    // sync to flush the buffer
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float handleBindingsTime;
    hipEventElapsedTime(&handleBindingsTime, start, stop);
    //printf("Successfully handled bindings\n");
    // Timing: Memory copy from device to host
    hipEventRecord(start);
    cudaStatus = hipMemcpy(molecules, dev_molecules, space->num_molecules * sizeof(Molecule), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for dev_molecules to molecules!\n");
        return cudaStatus;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float memcpyFromDeviceTime;
    hipEventElapsedTime(&memcpyFromDeviceTime, start, stop);
    //printf("Successfully copied molecules from device to host\n");
    // Timing: Process creation and deletion flags
    t1 = std::chrono::high_resolution_clock::now();
    int h_numCreations, h_numDeletions;
    hipMemcpy(&h_numCreations, dev_numCreations, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_numDeletions, dev_numDeletions, sizeof(int), hipMemcpyDeviceToHost);

    MoleculeCreationInfo* h_creationBuffer = nullptr;
    int* h_deletionBuffer = nullptr;

    if (h_numCreations > 0) {
        h_creationBuffer = new MoleculeCreationInfo[h_numCreations];
        hipMemcpy(h_creationBuffer, dev_creationBuffer, h_numCreations * sizeof(MoleculeCreationInfo), hipMemcpyDeviceToHost);
    }

    if (h_numDeletions > 0) {
        h_deletionBuffer = new int[h_numDeletions];
        hipMemcpy(h_deletionBuffer, dev_deletionBuffer, h_numDeletions * sizeof(int), hipMemcpyDeviceToHost);
    }

    //printf("Successfully first copied creation and deletion buffers from device to host\n");

    processCreationDeletionFlags(molecules, &space->num_molecules, MAX_MOLECULES,
                                 h_creationBuffer, h_numCreations,
                                 h_deletionBuffer, h_numDeletions);


    t2 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> processFlagsTime = t2 - t1;
    //printf("Successfully processed creation and deletion flags\n");
    // Reset the creation and deletion buffers
    hipMemset(dev_numCreations, 0, sizeof(int));
    hipMemset(dev_numDeletions, 0, sizeof(int));

    //printf("Successfully reset creation and deletion buffer counters\n");

    // Timing: copy molecules back to device
    hipEventRecord(start);
    cudaStatus = hipMemcpy(dev_molecules, molecules, space->num_molecules * sizeof(Molecule), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for molecules to dev_molecules!\n");
        return cudaStatus;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float secondMemcpyToDeviceTime;
    hipEventElapsedTime(&secondMemcpyToDeviceTime, start, stop);
    //printf("Successfully copied molecules back to device\n");
    // Timing: handle reactions and dissociations kernel
    t1 = std::chrono::high_resolution_clock::now();
    handleReactionsAndDissociations<<<blocksPerGrid, threadsPerBlock>>>(dev_molecules, dev_num_molecules, MAX_MOLECULES, dev_states,
                                                                     dev_creationBuffer, dev_numCreations,
                                                                     dev_deletionBuffer, dev_numDeletions);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float handleReactionsTime;
    hipEventElapsedTime(&handleReactionsTime, start, stop);
    //printf("Successfully handled reactions and dissociations\n");
    // Timing: copy molecules back to host
    hipEventRecord(start);
    cudaStatus = hipMemcpy(molecules, dev_molecules, space->num_molecules * sizeof(Molecule), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for dev_molecules to molecules!\n");
        return cudaStatus;
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float secondMemcpyFromDeviceTime;
    hipEventElapsedTime(&secondMemcpyFromDeviceTime, start, stop);
    //printf("Successfully copied molecules back to host\n");
    // Timing: process creation and deletion flags again
    t1 = std::chrono::high_resolution_clock::now();
    hipMemcpy(&h_numCreations, dev_numCreations, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_numDeletions, dev_numDeletions, sizeof(int), hipMemcpyDeviceToHost);

    //printf("Successfully copied creation and deletion buffers from device to host\n");

    if (h_numCreations > 0) {
        h_creationBuffer = new MoleculeCreationInfo[h_numCreations];
        hipMemcpy(h_creationBuffer, dev_creationBuffer, h_numCreations * sizeof(MoleculeCreationInfo), hipMemcpyDeviceToHost);
    }

    if (h_numDeletions > 0) {
        h_deletionBuffer = new int[h_numDeletions];
        hipMemcpy(h_deletionBuffer, dev_deletionBuffer, h_numDeletions * sizeof(int), hipMemcpyDeviceToHost);
    }

    //printf("Successfully second copied creation and deletion buffers from device to host\n");

    processCreationDeletionFlags(molecules, &space->num_molecules, MAX_MOLECULES,
                                 h_creationBuffer, h_numCreations,
                                 h_deletionBuffer, h_numDeletions);

    if (h_creationBuffer) delete[] h_creationBuffer;
    if (h_deletionBuffer) delete[] h_deletionBuffer;

    t2 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> secondProcessFlagsTime = t2 - t1;

    //printf("Successfully second processed creation and deletion flags\n");

    // Print timing results if enabled
    if (printTimings) {
        printf("Simulation Step Timings:\n");
        printf("  Memory Allocation: %.3f ms\n", memoryAllocationTime.count());
        printf("  Reset Cells: %.3f ms\n", resetCellsTime);
        if (initCurandTime > 0) printf("  Initialize cuRand: %.3f ms\n", initCurandTime);
        printf("  Memory Copy to Device: %.3f ms\n", memcpyToDeviceTime);
        printf("  Assign Molecules to Cells: %.3f ms\n", assignMoleculesTime);
        printf("  Apply Forces and Update Positions: %.3f ms\n", applyForcesTime);
        printf("  Handle Bindings: %.3f ms\n", handleBindingsTime);
        printf("  Memory Copy from Device: %.3f ms\n", memcpyFromDeviceTime);
        printf("  Process Creation/Deletion Flags: %.3f ms\n", processFlagsTime.count());
        printf("  Reset Creation/Deletion Buffers: %.3f ms\n", resetCreationDeletionBuffersTime);
        printf("  Memory Copy to Device: %.3f ms\n", secondMemcpyToDeviceTime);
        printf("  Handle Reactions and Dissociations: %.3f ms\n", handleReactionsTime);
        printf("  Memory Copy from Device: %.3f ms\n", secondMemcpyFromDeviceTime);
        printf("  Process Creation/Deletion Flags: %.3f ms\n", secondProcessFlagsTime.count());
        
        // Calculate and print total time
        float totalTime = memoryAllocationTime.count() + resetCellsTime + initCurandTime +
                          memcpyToDeviceTime + assignMoleculesTime + applyForcesTime +
                          handleBindingsTime + memcpyFromDeviceTime + processFlagsTime.count() +
                          resetCreationDeletionBuffersTime + secondMemcpyToDeviceTime +
                          handleReactionsTime + secondMemcpyFromDeviceTime + secondProcessFlagsTime.count();
        printf("  Total Calculated Time: %.3f ms\n", totalTime);
        printf("\n");
    }

    // Clean up CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    total_simulated_time += dt;

    return cudaStatus;

}

// Function to calculate MAX_MOLECULES_PER_CELL
int calculateMaxMoleculesPerCell(int max_molecules, float simulation_size, float cell_size) {
    int cells_per_axis = static_cast<int>(simulation_size / cell_size);
    int N_cells = cells_per_axis * cells_per_axis * cells_per_axis;

    float lambda = static_cast<float>(max_molecules) / static_cast<float>(N_cells);
    float sigma = sqrt(lambda);
    float expected_max = lambda + sigma * sqrt(2 * log(N_cells));

    // Choose a safety margin constant, e.g., k = 3
    float safety_margin = 3.0f * sigma;

    return static_cast<int>(expected_max + safety_margin + 0.5f);
}

// Main function
int main() {
    // Get and print CUDA device properties
    hipDeviceProp_t deviceProp;
    hipError_t cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed! Error: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    g_threadsToUse = deviceProp.maxThreadsPerBlock;
    // use the max number of blocks that can be launched per SM
    g_blocksToUse = deviceProp.maxBlocksPerMultiProcessor;
    //printf("Threads to use: %d\n", g_threadsToUse);
    //printf("Blocks to use: %d\n", g_blocksToUse);


    //printf("CUDA Device Properties:\n");
    //printf("  Device name: %s\n", deviceProp.name);
    //printf("  Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    //printf("  Total global memory: %zu bytes\n", deviceProp.totalGlobalMem);
    //printf("  Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    //printf("  Max threads dim: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    //printf("  Max grid size: (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    //printf("  Warp size: %d\n", deviceProp.warpSize);
    //printf("  Memory clock rate: %d kHz\n", deviceProp.memoryClockRate);
    //printf("  Memory bus width: %d bits\n", deviceProp.memoryBusWidth);
    //printf("\n");

    SimulationSpace space;
    Molecule* molecules = nullptr;

    // Read input from file
    FILE* input_file = fopen("input.txt", "r");
    if (input_file == NULL) {
        fprintf(stderr, "Failed to open input.txt\n");
        return 1;
    }

    char line[256];

    // Read simulation space dimensions
    while (fgets(line, sizeof(line), input_file)) {
        if (line[0] != '#') {
            int read_width, read_height, read_depth;
            if (sscanf(line, "%d %d %d", &read_width, &read_height, &read_depth) == 3) {
                space.width = (float)read_width;
                space.height = (float)read_height;
                space.depth = (float)read_depth;

                //printf("Read from file - Space width: %.2f\n", space.width);
                //printf("Read from file - Space height: %.2f\n", space.height);
                //printf("Read from file - Space depth: %.2f\n", space.depth);
            } else {
                fprintf(stderr, "Failed to read simulation space dimensions\n");
                // Set default values
                space.width = 100.0f;
                space.height = 100.0f;
                space.depth = 100.0f;
            }
            
            //printf("Simulation space dimensions: %.2f x %.2f x %.2f\n", space.width, space.height, space.depth);
            break;
        }
    }

    // Calculate max_molecules based on available memory
    size_t totalGlobalMem = deviceProp.totalGlobalMem;
    size_t moleculeSize = sizeof(Molecule);
    size_t moleculeCreationInfoSize = sizeof(MoleculeCreationInfo);
    size_t intSize = sizeof(int);

    // Reserve some memory for other data structures (e.g., 20%)
    size_t availableMemory = totalGlobalMem * 0.8;

    // Calculate max_molecules
    recommended_max_molecules = static_cast<int>(availableMemory / (moleculeSize + moleculeCreationInfoSize + intSize));

    // Calculate MAX_MOLECULES_PER_CELL
    recommended_max_molecules_per_cell = calculateMaxMoleculesPerCell(recommended_max_molecules, space.width, CELL_SIZE);

    printf("Recommended max molecules: %d\n", recommended_max_molecules);
    printf("Recommended max molecules per cell: %d\n", recommended_max_molecules_per_cell);
    printf("Molecule size: %zu bytes\n", moleculeSize);
    printf("Total global memory: %.2f GB\n", totalGlobalMem / 1e9);

    

    // Read number of molecule types
    while (fgets(line, sizeof(line), input_file)) {
        if (line[0] != '#') {
            if (sscanf(line, "%d", &space.num_molecule_types) != 1) {
                fprintf(stderr, "Failed to read number of molecule types\n");
                fclose(input_file);
                return 1;
            }
            //printf("Number of molecule types: %d\n", space.num_molecule_types);
            break;
        }
    }

    // Read molecule counts
    space.num_molecules = 0;
    int molecule_type = 0;
    while (fgets(line, sizeof(line), input_file) && molecule_type < space.num_molecule_types) {
        if (line[0] != '#') {
            char molecule_name[50];
            int count;
            if (sscanf(line, "%[^:]: %d", molecule_name, &count) == 2) {
                space.molecule_counts[molecule_type] = count;
                space.num_molecules += count;
                //printf("Molecule type %d (%s): %d\n", molecule_type, molecule_name, count);
                molecule_type++;
            }
        }
    }

    fclose(input_file);

    //printf("Total number of molecules: %d\n", space.num_molecules);

    // Before allocating memory for molecules
    if (space.num_molecules > MAX_MOLECULES) {
        fprintf(stderr, "Error: Number of molecules (%d) exceeds maximum allowed (%d)\n", space.num_molecules, MAX_MOLECULES);
        return 1;
    }

    // Allocate memory for molecules
    molecules = (Molecule*)malloc(MAX_MOLECULES * sizeof(Molecule));
    assert(molecules != nullptr); // Ensure memory allocation was successful

    //printf("Molecules allocated successfully\n");

    // Initialize molecules
    int molecule_index = 0;
    int type_counts[MAX_MOLECULE_TYPES] = {0};  // Array to keep count of each molecule type

    for (int i = 0; i < space.num_molecule_types; i++) {
        MoleculeType currentType = static_cast<MoleculeType>(i);
        int count = space.molecule_counts[i];

        //printf("Creating %d molecules of type %d (%s)\n", count, i, getMoleculeTypeName(currentType));

        for (int j = 0; j < count; j++) {
            if (molecule_index >= MAX_MOLECULES) {
                fprintf(stderr, "Error: Exceeded maximum number of molecules (%d)\n", MAX_MOLECULES);
                free(molecules);
                return 1;
            }

            Molecule newMol = createMolecule(currentType);

            // Initialize position randomly
            float x = static_cast<float>(rand()) / RAND_MAX * space.width;
            float y = static_cast<float>(rand()) / RAND_MAX * space.height;
            float z = static_cast<float>(rand()) / RAND_MAX * space.depth;
            newMol.centerOfMass = make_float3(x, y, z);

            molecules[molecule_index++] = newMol;

            type_counts[i]++;
        }
        //printf("Created molecule type %d (%s): %d\n", i, getMoleculeTypeName(currentType), type_counts[i]);
    }

    // Print summary of all molecule types created
    //printf("\nSummary of molecules created:\n");
    for (int i = 0; i < space.num_molecule_types; i++) {
        if (type_counts[i] > 0) {
            //printf("%s: %d\n", getMoleculeTypeName(static_cast<MoleculeType>(i)), type_counts[i]);
        }
    }

    // Check if we've initialized the correct number of molecules
    if (molecule_index != space.num_molecules) {
        fprintf(stderr, "Error: Initialized %d molecules, expected %d\n", molecule_index, space.num_molecules);
        free(molecules);
        return 1;
    }

    //printf("\nTotal molecules initialized successfully: %d\n", molecule_index);

    // Initialize visualization
    initVisualization();

    std::chrono::high_resolution_clock::time_point simulationStepStart, simulationStepEnd;

    // Main simulation loop
    std::chrono::high_resolution_clock::time_point loopStart, loopEnd, renderStart, renderEnd;
    while (!glfwWindowShouldClose(window)) {
        loopStart = std::chrono::high_resolution_clock::now();
        simulationStepStart = std::chrono::high_resolution_clock::now();

        if (!isPaused) {
            // Run a single step of the simulation
            cudaStatus = runSimulationStep(&space, molecules);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "Simulation step failed! Error: %s\n", hipGetErrorString(cudaStatus));
                break;
            }
        }

        simulationStepEnd = std::chrono::high_resolution_clock::now();

        renderStart = std::chrono::high_resolution_clock::now();
        if (!isRenderingPaused) {
            // Render the current state of the simulation
            renderSimulation(space, std::vector<Molecule>(molecules, molecules + space.num_molecules), total_simulated_time, 1.0f);
        }
        else {
            if (_kbhit()) {  // Check if a key has been pressed
                char ch = _getch();  // Get the pressed key
                if (ch == 'r' || ch == 'R') {  // Check if the key is 'r' or 'R'
                    printf("Rendering %s\n", isRenderingPaused ? "paused" : "resumed");
                    isRenderingPaused = !isRenderingPaused;  // Toggle rendering pause state
                    //printf("Rendering %s\n", isRenderingPaused ? "paused" : "resumed");
                }
                if (ch == 'w' || ch == 'W') {
                    // Write the current state of the simulation to a file
                    FILE* output_file = fopen("output.txt", "w");
                    if (output_file == NULL) {
                        fprintf(stderr, "Failed to open output.txt\n");
                        break;
                    }
                    // Write the current state of the simulation to the file
                    fprintf(output_file, "Current time: %.2f seconds\n", total_simulated_time);
                    // Write the count of each molecule type
                    for (int i = 0; i < space.num_molecule_types; i++) {
                        fprintf(output_file, "%s: %d\n", getMoleculeTypeName(static_cast<MoleculeType>(i)), space.molecule_counts[i]);
                    }
                    fclose(output_file);
                    printf("Simulation state written to output.txt\n");
                    printf("Press the space bar to continue...\n");

                    // Wait for the space bar to be pressed again before continuing
                    while (_getch() != 32) {  // 32 is the ASCII code for space bar
                        clock_t start_time = clock();
                        while (clock() - start_time < 100000); // Approximately 100ms delay
                    }
                }
                if (ch == 'l' || ch == 'L') {
                    // turn on/off printTimings
                    printTimings = !printTimings;
                    printf("PrintTimings %s\n", printTimings ? "enabled" : "disabled");
                }
            }
        }
        renderEnd = std::chrono::high_resolution_clock::now();

        loopEnd = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> loopTime = loopEnd - loopStart;
        std::chrono::duration<double, std::milli> simulationStepTime = simulationStepEnd - simulationStepStart;
        std::chrono::duration<double, std::milli> renderTime = renderEnd - renderStart;
        if (!isPaused && printTimings) {
            printf("Total Loop Time: %.3f ms\n", loopTime.count());
        }
        if (!isPaused && printTimings) {
            printf("Simulation Step Time: %.3f ms\n", simulationStepTime.count());
        }
        if (!isPaused && printTimings) {
            printf("Render Time: %.3f ms\n", renderTime.count());
        }

        // wait for key press to continue
        //printf("Press any key to continue...\n");
        //_getch();

    }



    // Cleanup
    cleanupVisualization();
    free(molecules);

    return 0;
}