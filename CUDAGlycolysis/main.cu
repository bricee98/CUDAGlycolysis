#include "hip/hip_runtime.h"
﻿#include <GL/glew.h>
#include <GL/gl.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "SimulationSpace.h"
#include "Molecule.cuh"
#include "kernel.cuh"
#include "visualization.h"
#include "Cell.cuh"
#include "SimulationData.h"



// Define constants
#define MAX_MOLECULES 2000
#define MAX_MOLECULE_TYPES 33
#define NUM_REACTION_TYPES 10 // Update this as you add more reaction types

// Constants for force calculations
#define COULOMB_CONSTANT 8.99e9f  // N*m^2/C^2
#define CUTOFF_DISTANCE 2.0f      // nm
#define CUTOFF_DISTANCE_SQ (CUTOFF_DISTANCE * CUTOFF_DISTANCE)
#define EPSILON_0 8.854187817e-12f // Vacuum permittivity
#define K_BOLTZMANN 1.380649e-23f  // Boltzmann constant
#define TEMPERATURE 310.15f        // Temperature in Kelvin (37°C)
#define SOLVENT_DIELECTRIC 78.5f   // Dielectric constant of water at 37°C

// Update these constants at the top of the file
#define MAX_THREADS_PER_BLOCK 1024
#define MAX_BLOCKS 65535

// Add these declarations
extern int h_GRID_SIZE_X;
extern int h_GRID_SIZE_Y;
extern int h_GRID_SIZE_Z;

// Define window and isPaused
GLFWwindow* window;
bool isPaused = false;

// Function prototypes
hipError_t runSimulation(SimulationSpace* space, Molecule* molecules, int num_ticks);
hipError_t runSimulationStep(SimulationSpace* space, Molecule* molecules);

// Add this function before the main() function

const char* getMoleculeTypeName(MoleculeType type) {
    switch(type) {
        case GLUCOSE: return "GLUCOSE";
        case ATP: return "ATP";
        case ADP: return "ADP";
        case GLUCOSE_6_PHOSPHATE: return "GLUCOSE_6_PHOSPHATE";
        case FRUCTOSE_6_PHOSPHATE: return "FRUCTOSE_6_PHOSPHATE";
        case FRUCTOSE_1_6_BISPHOSPHATE: return "FRUCTOSE_1_6_BISPHOSPHATE";
        case DIHYDROXYACETONE_PHOSPHATE: return "DIHYDROXYACETONE_PHOSPHATE";
        case GLYCERALDEHYDE_3_PHOSPHATE: return "GLYCERALDEHYDE_3_PHOSPHATE";
        case _1_3_BISPHOSPHOGLYCERATE: return "1_3_BISPHOSPHOGLYCERATE";
        case _3_PHOSPHOGLYCERATE: return "3_PHOSPHOGLYCERATE";
        case _2_PHOSPHOGLYCERATE: return "2_PHOSPHOGLYCERATE";
        case PHOSPHOENOLPYRUVATE: return "PHOSPHOENOLPYRUVATE";
        case PYRUVATE: return "PYRUVATE";
        case NAD_PLUS: return "NAD_PLUS";
        case NADH: return "NADH";
        case PROTON: return "PROTON";
        case INORGANIC_PHOSPHATE: return "INORGANIC_PHOSPHATE";
        case WATER: return "WATER";
        case HEXOKINASE: return "HEXOKINASE";
        case GLUCOSE_6_PHOSPHATE_ISOMERASE: return "GLUCOSE_6_PHOSPHATE_ISOMERASE";
        case PHOSPHOFRUCTOKINASE_1: return "PHOSPHOFRUCTOKINASE_1";
        case ALDOLASE: return "ALDOLASE";
        case TRIOSEPHOSPHATE_ISOMERASE: return "TRIOSEPHOSPHATE_ISOMERASE";
        case GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE: return "GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE";
        case PHOSPHOGLYCERATE_KINASE: return "PHOSPHOGLYCERATE_KINASE";
        case PHOSPHOGLYCERATE_MUTASE: return "PHOSPHOGLYCERATE_MUTASE";
        case ENOLASE: return "ENOLASE";
        case PYRUVATE_KINASE: return "PYRUVATE_KINASE";
        case AMP: return "AMP";
        case CITRATE: return "CITRATE";
        case FRUCTOSE_2_6_BISPHOSPHATE: return "FRUCTOSE_2_6_BISPHOSPHATE";
        default: return "UNKNOWN";
    }
}

// Host function to create a molecule
Molecule createMolecule(MoleculeType type) {
    switch (type) {
        // Substrates and products
        case GLUCOSE:
            return Molecule::createGlucose();
        case ATP:
            return Molecule::createATP();
        case ADP:
            return Molecule::createADP();
        case GLUCOSE_6_PHOSPHATE:
            return Molecule::createGlucose6Phosphate();
        case FRUCTOSE_6_PHOSPHATE:
            return Molecule::createFructose6Phosphate();
        case FRUCTOSE_1_6_BISPHOSPHATE:
            return Molecule::createFructose16Bisphosphate();
        case DIHYDROXYACETONE_PHOSPHATE:
            return Molecule::createDihydroxyacetonePhosphate();
        case GLYCERALDEHYDE_3_PHOSPHATE:
            return Molecule::createGlyceraldehyde3Phosphate();
        case _1_3_BISPHOSPHOGLYCERATE:
            return Molecule::create13Bisphosphoglycerate();
        case _3_PHOSPHOGLYCERATE:
            return Molecule::create3Phosphoglycerate();
        case _2_PHOSPHOGLYCERATE:
            return Molecule::create2Phosphoglycerate();
        case PHOSPHOENOLPYRUVATE:
            return Molecule::createPhosphoenolpyruvate();
        case PYRUVATE:
            return Molecule::createPyruvate();
        case NAD_PLUS:
            return Molecule::createNADPlus();
        case NADH:
            return Molecule::createNADH();
        case PROTON:
            return Molecule::createProton();
        case INORGANIC_PHOSPHATE:
            return Molecule::createInorganicPhosphate();
        case WATER:
            return Molecule::createWater();

        // Enzymes
        case HEXOKINASE:
            return Molecule::createHexokinase();
        case GLUCOSE_6_PHOSPHATE_ISOMERASE:
            return Molecule::createGlucose6PhosphateIsomerase();
        case PHOSPHOFRUCTOKINASE_1:
            return Molecule::createPhosphofructokinase1();
        case ALDOLASE:
            return Molecule::createAldolase();
        case TRIOSEPHOSPHATE_ISOMERASE:
            return Molecule::createTriosephosphateIsomerase();
        case GLYCERALDEHYDE_3_PHOSPHATE_DEHYDROGENASE:
            return Molecule::createGlyceraldehyde3PhosphateDehydrogenase();
        case PHOSPHOGLYCERATE_KINASE:
            return Molecule::createPhosphoglycerateKinase();
        case PHOSPHOGLYCERATE_MUTASE:
            return Molecule::createPhosphoglycerateMutase();
        case ENOLASE:
            return Molecule::createEnolase();
        case PYRUVATE_KINASE:
            return Molecule::createPyruvateKinase();

        // Regulatory molecules
        case AMP:
            return Molecule::createAMP();
        case CITRATE:
            return Molecule::createCitrate();
        case FRUCTOSE_2_6_BISPHOSPHATE:
            return Molecule::createFructose26Bisphosphate();

        default:
            fprintf(stderr, "Unknown molecule type\n");
            return Molecule::createWater();  // Default case, could also throw an exception
    }
}

// Host function to process creation and deletion flags
void processCreationDeletionFlags(Molecule* molecules, int* num_molecules, int max_molecules,
                                  const MoleculeCreationInfo* creationBuffer, int numCreations,
                                  const int* deletionBuffer, int numDeletions) {
    // Process deletions
    if (deletionBuffer && numDeletions > 0) {
        for (int i = 0; i < numDeletions; i++) {
            int indexToDelete = deletionBuffer[i];
            if (indexToDelete < *num_molecules - 1) {
                molecules[indexToDelete] = molecules[*num_molecules - 1];
            }
            (*num_molecules)--;
        }
    }

    // Process creations
    if (creationBuffer && numCreations > 0) {
        for (int i = 0; i < numCreations; i++) {
            if (*num_molecules < max_molecules) {
                Molecule newMolecule = createMolecule(creationBuffer[i].type);
                newMolecule.centerOfMass = make_float3(creationBuffer[i].x, creationBuffer[i].y, creationBuffer[i].z);
                molecules[*num_molecules] = newMolecule;
                (*num_molecules)++;
            }
        }
    }
}

// Modify the runSimulation function to use runSimulationStep
hipError_t runSimulation(SimulationSpace* space, Molecule* molecules, int num_ticks) {
    printf("Starting simulation with %d molecules for %d ticks\n", space->num_molecules, num_ticks);

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }
    printf("CUDA device set successfully\n");

    // Main simulation loop
    for (int tick = 0; tick < num_ticks; tick++) {
        printf("Starting tick %d\n", tick);

        cudaStatus = runSimulationStep(space, molecules);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Simulation step failed! Error: %s\n", hipGetErrorString(cudaStatus));
            break;
        }

        printf("Completed tick %d\n", tick);
    }

    printf("Simulation completed\n");

    return cudaStatus;
}

hipError_t runSimulationStep(SimulationSpace* space, Molecule* molecules) {
    static Molecule* dev_molecules = nullptr;
    static float3* dev_forces = nullptr;
    static hiprandState* dev_states = nullptr;
    static int* dev_reactionCounts = nullptr;
    static int* dev_num_molecules = nullptr;
    static MoleculeCreationInfo* dev_creationBuffer = nullptr;
    static int* dev_numCreations = nullptr;
    static int* dev_deletionBuffer = nullptr;
    static int* dev_numDeletions = nullptr;
    static Cell* dev_cells = nullptr;

    // Remove the extern variables and define grid sizes locally
    int gridSizeX = static_cast<int>(space->width / CELL_SIZE);
    int gridSizeY = static_cast<int>(space->height / CELL_SIZE);
    int gridSizeZ = static_cast<int>(space->depth / CELL_SIZE);

    Grid grid;
    grid.sizeX = gridSizeX;
    grid.sizeY = gridSizeY;
    grid.sizeZ = gridSizeZ;

    printf("Grid Sizes - X: %d, Y: %d, Z: %d\n", grid.sizeX, grid.sizeY, grid.sizeZ);

    hipError_t cudaStatus;
    int threadsPerBlock = 256;
    int blocksPerGrid = min((space->num_molecules + threadsPerBlock - 1) / threadsPerBlock, MAX_BLOCKS);

    // Ensure blocksPerGrid is at least 1
    if (blocksPerGrid < 1) blocksPerGrid = 1;

    // Calculate total cells
    int totalCells = grid.sizeX * grid.sizeY * grid.sizeZ;
    printf("Total cells: %d\n", totalCells);

    if (dev_cells == nullptr) {
        cudaStatus = hipMalloc(&dev_cells, totalCells * sizeof(Cell));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed for dev_cells!\n");
            return cudaStatus;
        }
    }

    // Reset cells
    hipMemset(dev_cells, 0, totalCells * sizeof(Cell));

    // Initialize device memory if it's the first run
    if (dev_molecules == nullptr) {
        // Allocate GPU buffers and copy data
        cudaStatus = hipMalloc((void**)&dev_molecules, MAX_MOLECULES * sizeof(Molecule));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed for dev_molecules! Error: %s\n", hipGetErrorString(cudaStatus));
            return cudaStatus;
        }
        cudaStatus = hipMalloc((void**)&dev_forces, MAX_MOLECULES * sizeof(float3));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed for dev_forces!\n"); return cudaStatus; }
        cudaStatus = hipMalloc((void**)&dev_states, MAX_MOLECULES * sizeof(hiprandState));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed for dev_states!\n"); return cudaStatus; }
        cudaStatus = hipMalloc(&dev_reactionCounts, NUM_REACTION_TYPES * sizeof(int));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed for dev_reactionCounts!\n"); return cudaStatus; }
        cudaStatus = hipMalloc((void**)&dev_num_molecules, sizeof(int));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed for dev_num_molecules!\n"); return cudaStatus; }
        cudaStatus = hipMalloc((void**)&dev_creationBuffer, MAX_MOLECULES * sizeof(MoleculeCreationInfo));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed for dev_creationBuffer!\n"); return cudaStatus; }
        cudaStatus = hipMalloc((void**)&dev_numCreations, sizeof(int));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed for dev_numCreations!\n"); return cudaStatus; }
        cudaStatus = hipMalloc((void**)&dev_deletionBuffer, MAX_MOLECULES * sizeof(int));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed for dev_deletionBuffer!\n"); return cudaStatus; }
        cudaStatus = hipMalloc((void**)&dev_numDeletions, sizeof(int));
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed for dev_numDeletions!\n"); return cudaStatus; }

        // Initialize hiprandState
        initCurand<<<blocksPerGrid, threadsPerBlock>>>(time(NULL), dev_states, space->num_molecules);
    }


    // available memory
    size_t availableMemory;
    cudaStatus = hipMemGetInfo(&availableMemory, NULL);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemGetInfo failed! Error: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }
    printf("Available memory: %zu bytes\n", availableMemory);

    // Copy current state to device
    cudaStatus = hipMemcpy(dev_molecules, molecules, MAX_MOLECULES * sizeof(Molecule), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for molecules to dev_molecules! Error: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    printf("Copied molecules to device\n");

    cudaStatus = hipMemcpy(dev_num_molecules, &space->num_molecules, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed for num_molecules to dev_num_molecules!\n"); return cudaStatus; }
    printf("Copied num_molecules to device\n");

    hipMemset(dev_reactionCounts, 0, NUM_REACTION_TYPES * sizeof(int));
    hipMemset(dev_numCreations, 0, sizeof(int));
    hipMemset(dev_numDeletions, 0, sizeof(int));
    printf("Reset reaction counts, numCreations, and numDeletions\n");
    // Assign molecules to cells
    dim3 gridAssign((space->num_molecules + threadsPerBlock - 1) / threadsPerBlock);
    assignMoleculesToCells<<<gridAssign, threadsPerBlock>>>(dev_molecules, space->num_molecules, dev_cells, *space, grid);
    printf("Assigned molecules to cells\n");
    // Synchronize to ensure kernel execution completion and flush printf output
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching assignMoleculesToCells kernel!\n", cudaStatus);
        return cudaStatus;
    }


    // Reset forces
    hipMemset(dev_forces, 0, space->num_molecules * sizeof(float3));
    printf("Reset forces\n");
    // Adjust the timestep for microsecond timescales
    float dt = 1e-6f; // Timestep of 1 microsecond

    // Call applyForcesAndUpdatePositions with the new dt
    applyForcesAndUpdatePositions<<<blocksPerGrid, threadsPerBlock>>>(
        dev_molecules, space->num_molecules, *space, dt, dev_states);

    // Synchronize to ensure kernel execution completion and flush printf output
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching applyForcesAndUpdatePositions kernel!\n", cudaStatus);
        return cudaStatus;
    }
    printf("Applied forces and updated positions\n");
    // Handle interactions
    handleInteractions<<<blocksPerGrid, threadsPerBlock>>>(dev_molecules, dev_num_molecules, MAX_MOLECULES, dev_states,
                                                           dev_reactionCounts, dev_creationBuffer, dev_numCreations,
                                                           dev_deletionBuffer, dev_numDeletions);

    // Check for errors during kernel launch
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error after handleInteractions kernel launch: %s\n", hipGetErrorString(err));
        return err;
    }

    // Synchronize to ensure kernel execution completion and flush printf output
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching handleInteractions kernel!\n", cudaStatus);
        return cudaStatus;
    }

    // Copy the updated molecules back to the host
    cudaStatus = hipMemcpy(molecules, dev_molecules, space->num_molecules * sizeof(Molecule), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed for dev_molecules to molecules!\n");
        return cudaStatus;
    }

    printf("Copied updated molecules to host\n");

    printf("Handled interactions\n");
    // Copy the number of creations and deletions back to the host
    int h_numCreations, h_numDeletions;
    hipMemcpy(&h_numCreations, dev_numCreations, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_numDeletions, dev_numDeletions, sizeof(int), hipMemcpyDeviceToHost);
    printf("Copied numCreations (%d) and numDeletions (%d) to host\n", h_numCreations, h_numDeletions);

    // Allocate host buffers for creations and deletions
    MoleculeCreationInfo* h_creationBuffer = nullptr;
    int* h_deletionBuffer = nullptr;

    if (h_numCreations > 0) {
        h_creationBuffer = new MoleculeCreationInfo[h_numCreations];
        // Copy creation buffer back to the host
        hipMemcpy(h_creationBuffer, dev_creationBuffer, h_numCreations * sizeof(MoleculeCreationInfo), hipMemcpyDeviceToHost);
        printf("Allocated and copied creation buffer to host\n");
    }

    if (h_numDeletions > 0) {
        h_deletionBuffer = new int[h_numDeletions];
        // Copy deletion buffer back to the host
        hipMemcpy(h_deletionBuffer, dev_deletionBuffer, h_numDeletions * sizeof(int), hipMemcpyDeviceToHost);
        printf("Allocated and copied deletion buffer to host\n");
    }

    // Process creation and deletion flags
    processCreationDeletionFlags(molecules, &space->num_molecules, MAX_MOLECULES,
                                 h_creationBuffer, h_numCreations,
                                 h_deletionBuffer, h_numDeletions);
    printf("Processed creation and deletion flags\n");

    // Free host buffers
    if (h_creationBuffer) {
        delete[] h_creationBuffer;
    }
    if (h_deletionBuffer) {
        delete[] h_deletionBuffer;
    }
    printf("Freed host buffers\n");
    // Copy updated molecules back to the device
    cudaStatus = hipMemcpy(dev_molecules, molecules, space->num_molecules * sizeof(Molecule), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed for molecules to dev_molecules!\n"); return cudaStatus; }
    printf("Copied updated molecules to device\n");

    // Print the position of molecule 500
    printf("Host Molecule 500: Position (%f, %f, %f)\n", molecules[500].centerOfMass.x, molecules[500].centerOfMass.y, molecules[500].centerOfMass.z);
    printf("Host Molecule 500: Velocity (%f, %f, %f)\n", molecules[500].vx, molecules[500].vy, molecules[500].vz);

    cudaStatus = hipMemcpy(dev_num_molecules, &space->num_molecules, sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed for num_molecules to dev_num_molecules!\n"); return cudaStatus; }
    printf("Copied updated num_molecules to device\n");
    // Check for errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }
    printf("Checked for errors\n");
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernels!\n", cudaStatus);
        return cudaStatus;
    }
    printf("Synchronized device\n");
    // Copy updated molecules back to host
    cudaStatus = hipMemcpy(molecules, dev_molecules, space->num_molecules * sizeof(Molecule), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed for dev_molecules to molecules!\n"); return cudaStatus; }
    printf("Copied updated molecules to host\n");
    return cudaStatus;
}

// Main function
int main() {
    // Get and print CUDA device properties
    hipDeviceProp_t deviceProp;
    hipError_t cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed! Error: %s\n", hipGetErrorString(cudaStatus));
        return 1;
    }

    printf("CUDA Device Properties:\n");
    printf("  Device name: %s\n", deviceProp.name);
    printf("  Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("  Total global memory: %zu bytes\n", deviceProp.totalGlobalMem);
    printf("  Max threads per block: %d\n", deviceProp.maxThreadsPerBlock);
    printf("  Max threads dim: (%d, %d, %d)\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("  Max grid size: (%d, %d, %d)\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    printf("  Warp size: %d\n", deviceProp.warpSize);
    printf("  Memory clock rate: %d kHz\n", deviceProp.memoryClockRate);
    printf("  Memory bus width: %d bits\n", deviceProp.memoryBusWidth);
    printf("\n");

    SimulationSpace space;
    Molecule* molecules = nullptr;

    // Read input from file
    FILE* input_file = fopen("input.txt", "r");
    if (input_file == NULL) {
        fprintf(stderr, "Failed to open input.txt\n");
        return 1;
    }

    char line[256];

    // Read simulation space dimensions
    while (fgets(line, sizeof(line), input_file)) {
        if (line[0] != '#') {
            int read_width, read_height, read_depth;
            if (sscanf(line, "%d %d %d", &read_width, &read_height, &read_depth) == 3) {
                space.width = (float)read_width;
                space.height = (float)read_height;
                space.depth = (float)read_depth;

                printf("Read from file - Space width: %.2f\n", space.width);
                printf("Read from file - Space height: %.2f\n", space.height);
                printf("Read from file - Space depth: %.2f\n", space.depth);
            } else {
                fprintf(stderr, "Failed to read simulation space dimensions\n");
                // Set default values
                space.width = 100.0f;
                space.height = 100.0f;
                space.depth = 100.0f;
            }
            
            printf("Simulation space dimensions: %.2f x %.2f x %.2f\n", space.width, space.height, space.depth);
            break;
        }
    }

    // Read number of molecule types
    while (fgets(line, sizeof(line), input_file)) {
        if (line[0] != '#') {
            if (sscanf(line, "%d", &space.num_molecule_types) != 1) {
                fprintf(stderr, "Failed to read number of molecule types\n");
                fclose(input_file);
                return 1;
            }
            printf("Number of molecule types: %d\n", space.num_molecule_types);
            break;
        }
    }

    // Read molecule counts
    space.num_molecules = 0;
    int molecule_type = 0;
    while (fgets(line, sizeof(line), input_file) && molecule_type < space.num_molecule_types) {
        if (line[0] != '#') {
            char molecule_name[50];
            int count;
            if (sscanf(line, "%[^:]: %d", molecule_name, &count) == 2) {
                space.molecule_counts[molecule_type] = count;
                space.num_molecules += count;
                printf("Molecule type %d (%s): %d\n", molecule_type, molecule_name, count);
                molecule_type++;
            }
        }
    }

    fclose(input_file);

    printf("Total number of molecules: %d\n", space.num_molecules);

    // Before allocating memory for molecules
    if (space.num_molecules > MAX_MOLECULES) {
        fprintf(stderr, "Error: Number of molecules (%d) exceeds maximum allowed (%d)\n", space.num_molecules, MAX_MOLECULES);
        return 1;
    }

    // Allocate memory for molecules
    molecules = (Molecule*)malloc(MAX_MOLECULES * sizeof(Molecule));
    if (molecules == nullptr) {
        fprintf(stderr, "Failed to allocate memory for molecules\n");
        return 1;
    }

    printf("Molecules allocated successfully\n");

    // Initialize molecules
    int molecule_index = 0;
    int type_counts[MAX_MOLECULE_TYPES] = {0};  // Array to keep count of each molecule type

    for (int i = 0; i < space.num_molecule_types; i++) {
        MoleculeType currentType = static_cast<MoleculeType>(i);
        int count = space.molecule_counts[i];

        printf("Creating %d molecules of type %d (%s)\n", count, i, getMoleculeTypeName(currentType));

        for (int j = 0; j < count; j++) {
            if (molecule_index >= MAX_MOLECULES) {
                fprintf(stderr, "Error: Exceeded maximum number of molecules (%d)\n", MAX_MOLECULES);
                free(molecules);
                return 1;
            }

            Molecule newMol = createMolecule(currentType);

            // Initialize position randomly
            float x = static_cast<float>(rand()) / RAND_MAX * space.width;
            float y = static_cast<float>(rand()) / RAND_MAX * space.height;
            float z = static_cast<float>(rand()) / RAND_MAX * space.depth;
            newMol.centerOfMass = make_float3(x, y, z);

            molecules[molecule_index++] = newMol;

            type_counts[i]++;
        }
        printf("Created molecule type %d (%s): %d\n", i, getMoleculeTypeName(currentType), type_counts[i]);
    }

    // Print summary of all molecule types created
    printf("\nSummary of molecules created:\n");
    for (int i = 0; i < space.num_molecule_types; i++) {
        if (type_counts[i] > 0) {
            printf("%s: %d\n", getMoleculeTypeName(static_cast<MoleculeType>(i)), type_counts[i]);
        }
    }

    // Check if we've initialized the correct number of molecules
    if (molecule_index != space.num_molecules) {
        fprintf(stderr, "Error: Initialized %d molecules, expected %d\n", molecule_index, space.num_molecules);
        free(molecules);
        return 1;
    }

    printf("\nTotal molecules initialized successfully: %d\n", molecule_index);

    // Initialize visualization
    initVisualization();

    // Main simulation loop
    while (!glfwWindowShouldClose(window)) {
        if (!isPaused) {
            // Run a single step of the simulation
            cudaStatus = runSimulationStep(&space, molecules);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "Simulation step failed! Error: %s\n", hipGetErrorString(cudaStatus));
                break;
            }
        }

        // Render the current state of the simulation
        renderSimulation(space, std::vector<Molecule>(molecules, molecules + space.num_molecules));
    }

    // Cleanup
    cleanupVisualization();
    free(molecules);

    return 0;
}